#include <cblas.h>
#include <iostream>
#include <math.h>
#include <chrono>
#include <hip/hip_runtime.h>
#include <hipblas.h>

using namespace std;
using namespace std::chrono;

#define M 10000
#define K 1000
#define N 10000

int main()
{
    double *A = new double[M * K];
    double *B = new double[K * N];
    double *C = new double[M * N];

    for (int i = 0; i < M * K; i++)
    {
        A[i] = sin(i);
    }

    for (int i = 0; i < K*N; i++){
        B[i] = cos(i);
    }

    for (int i = 0; i < M * N; i++)
    {
        C[i] = 0.5;
    }

    hipblasStatus_t stat; 
    hipError_t cudaStat; 
    hipblasHandle_t handle;               // CUBLAS context

    double *d_A;
    double *d_B;
    double *d_C;
    cudaStat = hipMalloc((void **)&d_A, M * K * sizeof(*A));
    cudaStat = hipMalloc((void **)&d_B, K * N * sizeof(*B));
    cudaStat = hipMalloc((void **)&d_C, M * N * sizeof(*C));

    stat = hipblasCreate(&handle); // initialize CUBLAS context

    // copy matrices from the host to the device
    stat = hipblasSetMatrix(M, K, sizeof(*A), A, M, d_A, M); //a -> d_a
    stat = hipblasSetMatrix(K, N, sizeof(*B), B, K, d_B, K); //b -> d_b
    stat = hipblasSetMatrix(M, N, sizeof(*C), C, M, d_C, M); //c -> d_c

    auto startTime = high_resolution_clock::now();
    double alpha = 1.0;
    double beta = 1.0;
    stat = hipblasDgemm(handle, HIPBLAS_OP_N, HIPBLAS_OP_N, M, N, K, &alpha, d_A,
        M, d_B, K, &beta, d_C, M);
    hipDeviceSynchronize();
    auto endTime = high_resolution_clock::now();
    auto duration = duration_cast<microseconds>(endTime - startTime); 
    cout << "cblas计算用时" <<double(duration.count()) / 1000000 << "s" << endl;

    stat = hipblasGetMatrix(M, N, sizeof(*C), d_C, M, C, M); // cp d_c - >c
    hipFree(d_A);         // free device memory
    hipFree(d_B);         // free device memory
    hipFree(d_C);         // free device memory
    hipblasDestroy(handle); // destroy CUBLAS context
    free(A);
    free(B);
    free(C);
}